#include "hip/hip_runtime.h"
/*
 * CUDA kernel code for NCS.
 *
 * Important note!! This will only work correctly with 16 work items per work group!!
 * Why 16? This matches the 1D size of the 2D FFT.
 *
 * Hazen 08/19
 */

/*
 * License for the Limited memory BFGS (L-BFGS) solver code.
 *
 * Limited memory BFGS (L-BFGS).
 *
 * Copyright (c) 1990, Jorge Nocedal
 * Copyright (c) 2007-2010 Naoaki Okazaki
 * All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 */

#include <hip/hip_vector_types.h>

/* Threshold for handling negative values in the fit. */
#define FITMIN 1.0e-6f

/*
 * The problem size is (16*16)/4 or 256/4.
 *
 * This matches the size of the FFT so changing these would involve writing
 * a different size FFT, etc..
 */
#define ASIZE 16
#define PSIZE 64

/* L-BFGS solver parameters. */

#define C_1 1.0e-4f        /* Armijo rule/condition scaling value. */
#define EPSILON 1.0e-4f    /* Stopping point. */
#define M 8                /* Number of history points saved. Must be a power of 2. */
#define MAXITERS 200       /* Maximum number of iterations. */
#define MIN_STEP 1.0e-6f   /* Minimum step size. */
#define STEPM 0.5          /* Step size multiplier. */

/* Error status codes. */
#define SUCCESS 0
#define UNSTARTED -1
#define REACHED_MAXITERS -2
#define INCREASING_GRADIENT -3
#define MINIMUM_STEP -4
#define REACHED_MAXPRECISION -5

/****************
 * FFT functions.
 ****************/

// 4 point complex FFT
__device__ void fft4(float4 x_r, float4 x_c, float4 *y_r, float4 *y_c)
{
    float t1_r = x_r.x + x_r.z;
    float t1_c = x_c.x + x_c.z;
    
    float t2_r = x_r.x - x_r.z;
    float t2_c = x_c.x - x_c.z;

    float t3_r = x_r.y + x_r.w;
    float t3_c = x_c.y + x_c.w;

    float t4_r = x_r.y - x_r.w;
    float t4_c = x_c.y - x_c.w;
 
    y_r[0].x = t1_r + t3_r;
    y_r[0].y = t2_r + t4_c;
    y_r[0].z = t1_r - t3_r;
    y_r[0].w = t2_r - t4_c;

    y_c[0].x = t1_c + t3_c;
    y_c[0].y = t2_c - t4_r;
    y_c[0].z = t1_c - t3_c;
    y_c[0].w = t2_c + t4_r;
}


// 4 point complex IFFT
__device__ void ifft4(float4 x_r, float4 x_c, float4 *y_r, float4 *y_c)
{
    float t1_r = x_r.x + x_r.z;
    float t1_c = x_c.x + x_c.z;
    
    float t2_r = x_r.x - x_r.z;
    float t2_c = x_c.x - x_c.z;

    float t3_r = x_r.y + x_r.w;
    float t3_c = x_c.y + x_c.w;

    float t4_r = x_r.y - x_r.w;
    float t4_c = x_c.y - x_c.w;
 
    y_r[0].x = t1_r + t3_r;
    y_r[0].y = t2_r - t4_c;
    y_r[0].z = t1_r - t3_r;
    y_r[0].w = t2_r + t4_c;

    y_c[0].x = t1_c + t3_c;
    y_c[0].y = t2_c + t4_r;
    y_c[0].z = t1_c - t3_c;
    y_c[0].w = t2_c - t4_r;
    
    y_r[0] = y_r[0]*0.25f;
    y_c[0] = y_c[0]*0.25f;
}


// 8 point complex FFT.
__device__ void fft8(float4 *x_r, float4 *x_c, float4 *y_r, float4 *y_c)
{
    float4 t_r;
    float4 t_c;
    float4 f41_r;
    float4 f41_c;
    float4 f42_r;
    float4 f42_c;
     
    // 4 point FFT.
    t_r = make_float4(x_r[0].x, x_r[0].z, x_r[1].x, x_r[1].z);
    t_c = make_float4(x_c[0].x, x_c[0].z, x_c[1].x, x_c[1].z);    
    fft4(t_r, t_c, &f41_r, &f41_c);

    t_r = make_float4(x_r[0].y, x_r[0].w, x_r[1].y, x_r[1].w);
    t_c = make_float4(x_c[0].y, x_c[0].w, x_c[1].y, x_c[1].w);
    fft4(t_r, t_c, &f42_r, &f42_c);
    
    // Shift and add.
    float4 r1 = make_float4(1.0f, 7.07106781e-01f, 0.0f, -7.07106781e-01f);
    float4 c1 = make_float4(0.0f, 7.07106781e-01f, 1.0f,  7.07106781e-01f);
    y_r[0] = f41_r + f42_r * r1 + f42_c * c1;
    y_c[0] = f41_c + f42_c * r1 - f42_r * c1;
    y_r[1] = f41_r - f42_r * r1 - f42_c * c1;
    y_c[1] = f41_c - f42_c * r1 + f42_r * c1;    
}


// 8 point complex IFFT.
__device__ void ifft8(float4 *x_r, float4 *x_c, float4 *y_r, float4 *y_c)
{
    float4 t_r;
    float4 t_c;
    float4 f41_r;
    float4 f41_c;
    float4 f42_r;
    float4 f42_c;
     
    // 4 point IFFT.
    t_r = make_float4(x_r[0].x, x_r[0].z, x_r[1].x, x_r[1].z);
    t_c = make_float4(x_c[0].x, x_c[0].z, x_c[1].x, x_c[1].z);    
    ifft4(t_r, t_c, &f41_r, &f41_c);

    t_r = make_float4(x_r[0].y, x_r[0].w, x_r[1].y, x_r[1].w);
    t_c = make_float4(x_c[0].y, x_c[0].w, x_c[1].y, x_c[1].w);
    ifft4(t_r, t_c, &f42_r, &f42_c);
    
    // Shift and add.
    float4 r1 = make_float4(1.0f,  7.07106781e-01f,  0.0f, -7.07106781e-01f);
    float4 c1 = make_float4(0.0f, -7.07106781e-01f, -1.0f, -7.07106781e-01f);
    y_r[0] = f41_r + f42_r * r1 + f42_c * c1;
    y_c[0] = f41_c + f42_c * r1 - f42_r * c1;
    y_r[1] = f41_r - f42_r * r1 - f42_c * c1;
    y_c[1] = f41_c - f42_c * r1 + f42_r * c1;
    
    y_r[0] = y_r[0]*0.5f;
    y_c[0] = y_c[0]*0.5f;
    y_r[1] = y_r[1]*0.5f;
    y_c[1] = y_c[1]*0.5f;
}


// 16 point complex FFT.
__device__ void fft16(float4 *x_r, float4 *x_c, float4 *y_r, float4 *y_c)
{
    float4 t_r[2];
    float4 t_c[2];
    float4 f41_r[2];
    float4 f41_c[2];
    float4 f42_r[2];
    float4 f42_c[2];
     
    // 8 point FFT.
    t_r[0] = make_float4(x_r[0].x, x_r[0].z, x_r[1].x, x_r[1].z);
    t_c[0] = make_float4(x_c[0].x, x_c[0].z, x_c[1].x, x_c[1].z);    
    t_r[1] = make_float4(x_r[2].x, x_r[2].z, x_r[3].x, x_r[3].z);
    t_c[1] = make_float4(x_c[2].x, x_c[2].z, x_c[3].x, x_c[3].z);    
    fft8(t_r, t_c, f41_r, f41_c);

    t_r[0] = make_float4(x_r[0].y, x_r[0].w, x_r[1].y, x_r[1].w);
    t_c[0] = make_float4(x_c[0].y, x_c[0].w, x_c[1].y, x_c[1].w);
    t_r[1] = make_float4(x_r[2].y, x_r[2].w, x_r[3].y, x_r[3].w);
    t_c[1] = make_float4(x_c[2].y, x_c[2].w, x_c[3].y, x_c[3].w);
    fft8(t_r, t_c, f42_r, f42_c);
        
    // Shift and add.
    float4 r1 = make_float4(1.0f, 9.23879533e-01f, 7.07106781e-01f, 3.82683432e-01f);
    float4 c1 = make_float4(0.0f, 3.82683432e-01f, 7.07106781e-01f, 9.23879533e-01f);
    y_r[0] = f41_r[0] + f42_r[0] * r1 + f42_c[0] * c1;
    y_c[0] = f41_c[0] + f42_c[0] * r1 - f42_r[0] * c1;    
    y_r[2] = f41_r[0] - f42_r[0] * r1 - f42_c[0] * c1;
    y_c[2] = f41_c[0] - f42_c[0] * r1 + f42_r[0] * c1;    
    
    r1 = make_float4(0.0f, -3.82683432e-01f, -7.07106781e-01f, -9.23879533e-01f);
    c1 = make_float4(1.0f,  9.23879533e-01f,  7.07106781e-01f,  3.82683432e-01f);
    y_r[1] = f41_r[1] + f42_r[1] * r1 + f42_c[1] * c1;
    y_c[1] = f41_c[1] + f42_c[1] * r1 - f42_r[1] * c1;    
    y_r[3] = f41_r[1] - f42_r[1] * r1 - f42_c[1] * c1;
    y_c[3] = f41_c[1] - f42_c[1] * r1 + f42_r[1] * c1;    
}


// 16 point complex FFT (__local variable version).
__device__ void fft16_lvar(float4 *x_r, float4 *x_c, float4 *y_r, float4 *y_c)
{
    float4 t_r[2];
    float4 t_c[2];
    float4 f41_r[2];
    float4 f41_c[2];
    float4 f42_r[2];
    float4 f42_c[2];
     
    // 8 point FFT.
    t_r[0] = make_float4(x_r[0].x, x_r[0].z, x_r[1].x, x_r[1].z);
    t_c[0] = make_float4(x_c[0].x, x_c[0].z, x_c[1].x, x_c[1].z);    
    t_r[1] = make_float4(x_r[2].x, x_r[2].z, x_r[3].x, x_r[3].z);
    t_c[1] = make_float4(x_c[2].x, x_c[2].z, x_c[3].x, x_c[3].z);    
    fft8(t_r, t_c, f41_r, f41_c);

    t_r[0] = make_float4(x_r[0].y, x_r[0].w, x_r[1].y, x_r[1].w);
    t_c[0] = make_float4(x_c[0].y, x_c[0].w, x_c[1].y, x_c[1].w);
    t_r[1] = make_float4(x_r[2].y, x_r[2].w, x_r[3].y, x_r[3].w);
    t_c[1] = make_float4(x_c[2].y, x_c[2].w, x_c[3].y, x_c[3].w);
    fft8(t_r, t_c, f42_r, f42_c);
        
    // Shift and add.
    float4 r1 = make_float4(1.0f, 9.23879533e-01f, 7.07106781e-01f, 3.82683432e-01f);
    float4 c1 = make_float4(0.0f, 3.82683432e-01f, 7.07106781e-01f, 9.23879533e-01f);
    y_r[0] = f41_r[0] + f42_r[0] * r1 + f42_c[0] * c1;
    y_c[0] = f41_c[0] + f42_c[0] * r1 - f42_r[0] * c1;    
    y_r[2] = f41_r[0] - f42_r[0] * r1 - f42_c[0] * c1;
    y_c[2] = f41_c[0] - f42_c[0] * r1 + f42_r[0] * c1;    

    r1 = make_float4(0.0f, -3.82683432e-01f, -7.07106781e-01f, -9.23879533e-01f);
    c1 = make_float4(1.0f,  9.23879533e-01f,  7.07106781e-01f,  3.82683432e-01f);
    y_r[1] = f41_r[1] + f42_r[1] * r1 + f42_c[1] * c1;
    y_c[1] = f41_c[1] + f42_c[1] * r1 - f42_r[1] * c1;    
    y_r[3] = f41_r[1] - f42_r[1] * r1 - f42_c[1] * c1;
    y_c[3] = f41_c[1] - f42_c[1] * r1 + f42_r[1] * c1;    
}


// 16 point complex IFFT.
__device__ void ifft16(float4 *x_r, float4 *x_c, float4 *y_r, float4 *y_c)
{
    float4 t_r[2];
    float4 t_c[2];
    float4 f41_r[2];
    float4 f41_c[2];
    float4 f42_r[2];
    float4 f42_c[2];
     
    // 8 point IFFT.
    t_r[0] = make_float4(x_r[0].x, x_r[0].z, x_r[1].x, x_r[1].z);
    t_c[0] = make_float4(x_c[0].x, x_c[0].z, x_c[1].x, x_c[1].z);    
    t_r[1] = make_float4(x_r[2].x, x_r[2].z, x_r[3].x, x_r[3].z);
    t_c[1] = make_float4(x_c[2].x, x_c[2].z, x_c[3].x, x_c[3].z);    
    ifft8(t_r, t_c, f41_r, f41_c);

    t_r[0] = make_float4(x_r[0].y, x_r[0].w, x_r[1].y, x_r[1].w);
    t_c[0] = make_float4(x_c[0].y, x_c[0].w, x_c[1].y, x_c[1].w);
    t_r[1] = make_float4(x_r[2].y, x_r[2].w, x_r[3].y, x_r[3].w);
    t_c[1] = make_float4(x_c[2].y, x_c[2].w, x_c[3].y, x_c[3].w);
    ifft8(t_r, t_c, f42_r, f42_c);
        
    // Shift and add.
    float4 r1 = make_float4(1.0f,  9.23879533e-01f,  7.07106781e-01f,  3.82683432e-01f);
    float4 c1 = make_float4(0.0f, -3.82683432e-01f, -7.07106781e-01f, -9.23879533e-01f);
    y_r[0] = f41_r[0] + f42_r[0] * r1 + f42_c[0] * c1;
    y_c[0] = f41_c[0] + f42_c[0] * r1 - f42_r[0] * c1;    
    y_r[2] = f41_r[0] - f42_r[0] * r1 - f42_c[0] * c1;
    y_c[2] = f41_c[0] - f42_c[0] * r1 + f42_r[0] * c1;    
    
    r1 = make_float4( 0.0f, -3.82683432e-01f, -7.07106781e-01f, -9.23879533e-01f);
    c1 = make_float4(-1.0f, -9.23879533e-01f, -7.07106781e-01f, -3.82683432e-01f);
    y_r[1] = f41_r[1] + f42_r[1] * r1 + f42_c[1] * c1;
    y_c[1] = f41_c[1] + f42_c[1] * r1 - f42_r[1] * c1;    
    y_r[3] = f41_r[1] - f42_r[1] * r1 - f42_c[1] * c1;
    y_c[3] = f41_c[1] - f42_c[1] * r1 + f42_r[1] * c1;
    
    for(int i = 0; i<4; i++){
        y_r[i] = y_r[i]*0.5f;
        y_c[i] = y_c[i]*0.5f;
    }
}


// 16 point complex IFFT (__local variable version).
__device__ void ifft16_lvar(float4 *x_r, float4 *x_c, float4 *y_r, float4 *y_c)
{
    float4 t_r[2];
    float4 t_c[2];
    float4 f41_r[2];
    float4 f41_c[2];
    float4 f42_r[2];
    float4 f42_c[2];
     
    // 8 point IFFT.
    t_r[0] = make_float4(x_r[0].x, x_r[0].z, x_r[1].x, x_r[1].z);
    t_c[0] = make_float4(x_c[0].x, x_c[0].z, x_c[1].x, x_c[1].z);    
    t_r[1] = make_float4(x_r[2].x, x_r[2].z, x_r[3].x, x_r[3].z);
    t_c[1] = make_float4(x_c[2].x, x_c[2].z, x_c[3].x, x_c[3].z);    
    ifft8(t_r, t_c, f41_r, f41_c);

    t_r[0] = make_float4(x_r[0].y, x_r[0].w, x_r[1].y, x_r[1].w);
    t_c[0] = make_float4(x_c[0].y, x_c[0].w, x_c[1].y, x_c[1].w);
    t_r[1] = make_float4(x_r[2].y, x_r[2].w, x_r[3].y, x_r[3].w);
    t_c[1] = make_float4(x_c[2].y, x_c[2].w, x_c[3].y, x_c[3].w);
    ifft8(t_r, t_c, f42_r, f42_c);
        
    // Shift and add.
    float4 r1 = make_float4(1.0f,  9.23879533e-01f,  7.07106781e-01f,  3.82683432e-01f);
    float4 c1 = make_float4(0.0f, -3.82683432e-01f, -7.07106781e-01f, -9.23879533e-01f);
    y_r[0] = f41_r[0] + f42_r[0] * r1 + f42_c[0] * c1;
    y_c[0] = f41_c[0] + f42_c[0] * r1 - f42_r[0] * c1;    
    y_r[2] = f41_r[0] - f42_r[0] * r1 - f42_c[0] * c1;
    y_c[2] = f41_c[0] - f42_c[0] * r1 + f42_r[0] * c1;    
    
    r1 = make_float4( 0.0f, -3.82683432e-01f, -7.07106781e-01f, -9.23879533e-01f);
    c1 = make_float4(-1.0f, -9.23879533e-01f, -7.07106781e-01f, -3.82683432e-01f);
    y_r[1] = f41_r[1] + f42_r[1] * r1 + f42_c[1] * c1;
    y_c[1] = f41_c[1] + f42_c[1] * r1 - f42_r[1] * c1;    
    y_r[3] = f41_r[1] - f42_r[1] * r1 - f42_c[1] * c1;
    y_c[3] = f41_c[1] - f42_c[1] * r1 + f42_r[1] * c1;
    
    for(int i = 0; i<4; i++){
        y_r[i] = y_r[i]*0.5f;
        y_c[i] = y_c[i]*0.5f;    
    }
}


// 16 x 16 point complex FFT with work group size of 16.
__device__ void fft_16x16_wg16(float4 *x_r, float4 *x_c, float4 *y_r, float4 *y_c, int lid)
{
    int j;
    
    float4 t1_r[4];
    float4 t1_c[4];
    
    float *y1_r = (float *)y_r;
    float *y1_c = (float *)y_c;
    
    // Axis 1.
    fft16_lvar(&(x_r[lid*4]), &(x_c[lid*4]), &(y_r[lid*4]), &(y_c[lid*4]));
    
    __syncthreads();
    
    // Axis 2.
    
    // Convert columns to rows.
    for(j=0; j<4; j++){
        t1_r[j].x = y1_r[(4*j+0)*16+lid];
        t1_r[j].y = y1_r[(4*j+1)*16+lid];
        t1_r[j].z = y1_r[(4*j+2)*16+lid];
        t1_r[j].w = y1_r[(4*j+3)*16+lid];
        t1_c[j].x = y1_c[(4*j+0)*16+lid];
        t1_c[j].y = y1_c[(4*j+1)*16+lid];
        t1_c[j].z = y1_c[(4*j+2)*16+lid];
        t1_c[j].w = y1_c[(4*j+3)*16+lid];    
    }
        
    fft16(t1_r, t1_c, t1_r, t1_c);
        
    // Reverse conversion.
    for(j=0; j<4; j++){
        y1_r[(4*j+0)*16+lid] = t1_r[j].x;
        y1_r[(4*j+1)*16+lid] = t1_r[j].y;
        y1_r[(4*j+2)*16+lid] = t1_r[j].z;
        y1_r[(4*j+3)*16+lid] = t1_r[j].w;
        y1_c[(4*j+0)*16+lid] = t1_c[j].x;
        y1_c[(4*j+1)*16+lid] = t1_c[j].y;
        y1_c[(4*j+2)*16+lid] = t1_c[j].z;
        y1_c[(4*j+3)*16+lid] = t1_c[j].w;            
    }
    
    __syncthreads();
}


// 16 x 16 point complex IFFT with work group size of 16.
__device__ void ifft_16x16_wg16(float4 *x_r, float4 *x_c, float4 *y_r, float4 *y_c, int lid)
{
    int j;
    
    float4 t1_r[4];
    float4 t1_c[4];

    float *x1_r = (float *)x_r;
    float *x1_c = (float *)x_c;
    float *y1_r = (float *)y_r;
    float *y1_c = (float *)y_c;
    
    // Axis 2.
    
    // Convert columns to rows.
    for(j=0; j<4; j++){
        t1_r[j].x = x1_r[(4*j+0)*16+lid];
        t1_r[j].y = x1_r[(4*j+1)*16+lid];
        t1_r[j].z = x1_r[(4*j+2)*16+lid];
        t1_r[j].w = x1_r[(4*j+3)*16+lid];
        t1_c[j].x = x1_c[(4*j+0)*16+lid];
        t1_c[j].y = x1_c[(4*j+1)*16+lid];
        t1_c[j].z = x1_c[(4*j+2)*16+lid];
        t1_c[j].w = x1_c[(4*j+3)*16+lid];    
     }
        
     ifft16(t1_r, t1_c, t1_r, t1_c);
        
     // Reverse conversion.
     for(j=0; j<4; j++){
         y1_r[(4*j+0)*16+lid] = t1_r[j].x;
         y1_r[(4*j+1)*16+lid] = t1_r[j].y;
         y1_r[(4*j+2)*16+lid] = t1_r[j].z;
         y1_r[(4*j+3)*16+lid] = t1_r[j].w;
         y1_c[(4*j+0)*16+lid] = t1_c[j].x;
         y1_c[(4*j+1)*16+lid] = t1_c[j].y;
         y1_c[(4*j+2)*16+lid] = t1_c[j].z;
         y1_c[(4*j+3)*16+lid] = t1_c[j].w;        
    }
    
    __syncthreads();
    
    // Axis 1.
    ifft16_lvar(&(y_r[lid*4]), &(y_c[lid*4]), &(y_r[lid*4]), &(y_c[lid*4]));

    __syncthreads();
}


/******************
 * Vector functions.
 *
 * These are all designed on vectors with 64 float4 elements
 * and lid values in the range (0 - 15).
 *
 ******************/

__device__ void veccopy(float4 *v1, float4 *v2, int lid)
{
    int i = lid*4;

    v1[i]   = v2[i];
    v1[i+1] = v2[i+1];
    v1[i+2] = v2[i+2];
    v1[i+3] = v2[i+3];
}

__device__ void vecncopy(float4 *v1, float4 *v2, int lid)
{
    int i = lid*4;

    v1[i]   = -v2[i];
    v1[i+1] = -v2[i+1];
    v1[i+2] = -v2[i+2];
    v1[i+3] = -v2[i+3];
}

/* Returns the dot product as the first element of w1. */
__device__ void vecdot(float *w1, float4 *v1, float4 *v2, int lid)
{
    int i = lid*4;
    float sum = 0.0f;

    sum += dot(v1[i]  , v2[i]);
    sum += dot(v1[i+1], v2[i+1]);
    sum += dot(v1[i+2], v2[i+2]);
    sum += dot(v1[i+3], v2[i+3]);
    w1[lid] = sum;

    __syncthreads();

    if(lid == 0){
    	for(i=1; i<16; i++){
	   w1[0] += w1[i];
	}
    }

    __syncthreads();
}

/* Returns 0 or a positive integer as the first element of w1. */
__device__ void vecisEqual(int *w1, float4 *v1, float4 *v2, int lid)
{
    int i = lid*4;
    int sum = 0;

    sum += (v1[i].x   != v2[i].x)   + (v1[i].y   != v2[i].y)   + (v1[i].z   != v2[i].z)   + (v1[i].w   != v2[i].w);
    sum += (v1[i+1].x != v2[i+1].x) + (v1[i+1].y != v2[i+1].y) + (v1[i+1].z != v2[i+1].z) + (v1[i+1].w != v2[i+1].w);
    sum += (v1[i+2].x != v2[i+2].x) + (v1[i+2].y != v2[i+2].y) + (v1[i+2].z != v2[i+2].z) + (v1[i+2].w != v2[i+2].w);
    sum += (v1[i+3].x != v2[i+3].x) + (v1[i+3].y != v2[i+3].y) + (v1[i+3].z != v2[i+3].z) + (v1[i+3].w != v2[i+3].w);
    w1[lid] = sum;

    __syncthreads();

    if(lid == 0){
    	for(i=1; i<16; i++){
	   w1[0] += w1[i];
	}
	w1[0] = !w1[0];
    }

    __syncthreads();
}

/* v1 = v2 * s1 + v3 */
__device__ void vecfma(float4 *v1, float4 *v2, float4 *v3, float s1, int lid)
{
    int i = lid*4;

    v1[i].x   = fma(s1, v2[i].x, v3[i].x);
    v1[i].y   = fma(s1, v2[i].y, v3[i].y);
    v1[i].z   = fma(s1, v2[i].z, v3[i].z);
    v1[i].w   = fma(s1, v2[i].w, v3[i].w);
    i += 1;

    v1[i].x   = fma(s1, v2[i].x, v3[i].x);
    v1[i].y   = fma(s1, v2[i].y, v3[i].y);
    v1[i].z   = fma(s1, v2[i].z, v3[i].z);
    v1[i].w   = fma(s1, v2[i].w, v3[i].w);
    i += 1;

    v1[i].x   = fma(s1, v2[i].x, v3[i].x);
    v1[i].y   = fma(s1, v2[i].y, v3[i].y);
    v1[i].z   = fma(s1, v2[i].z, v3[i].z);
    v1[i].w   = fma(s1, v2[i].w, v3[i].w);
    i += 1;

    v1[i].x   = fma(s1, v2[i].x, v3[i].x);
    v1[i].y   = fma(s1, v2[i].y, v3[i].y);
    v1[i].z   = fma(s1, v2[i].z, v3[i].z);
    v1[i].w   = fma(s1, v2[i].w, v3[i].w);
}

/* v1 = v2 * s1 + v1 */
__device__ void vecfmaInplace(float4 *v1, float4 *v2, float s1, int lid)
{
    int i = lid*4;

    v1[i].x   = fma(s1, v2[i].x, v1[i].x);
    v1[i].y   = fma(s1, v2[i].y, v1[i].y);
    v1[i].z   = fma(s1, v2[i].z, v1[i].z);
    v1[i].w   = fma(s1, v2[i].w, v1[i].w);
    i += 1;

    v1[i].x   = fma(s1, v2[i].x, v1[i].x);
    v1[i].y   = fma(s1, v2[i].y, v1[i].y);
    v1[i].z   = fma(s1, v2[i].z, v1[i].z);
    v1[i].w   = fma(s1, v2[i].w, v1[i].w);
    i += 1;

    v1[i].x   = fma(s1, v2[i].x, v1[i].x);
    v1[i].y   = fma(s1, v2[i].y, v1[i].y);
    v1[i].z   = fma(s1, v2[i].z, v1[i].z);
    v1[i].w   = fma(s1, v2[i].w, v1[i].w);
    i += 1;

    v1[i].x   = fma(s1, v2[i].x, v1[i].x);
    v1[i].y   = fma(s1, v2[i].y, v1[i].y);
    v1[i].z   = fma(s1, v2[i].z, v1[i].z);
    v1[i].w   = fma(s1, v2[i].w, v1[i].w);
}

__device__ void vecnorm(float *w1, float4 *v1, int lid)
{
    vecdot(w1, v1, v1, lid);

    if(lid == 0){
        w1[0] = sqrt(w1[0]);
    }

    __syncthreads();
}

__device__ void vecscaleInplace(float4 *v1, float s1, int lid)
{
    int i = lid*4;
    
    float4 t1 = make_float4(s1, s1, s1, s1);
    v1[i]   = v1[i]*t1;
    v1[i+1] = v1[i+1]*t1;
    v1[i+2] = v1[i+2]*t1;
    v1[i+3] = v1[i+3]*t1;
}

__device__ void vecsub(float4 *v1, float4 *v2, float4 *v3, int lid)
{
    int i = lid*4;
    
    v1[i]   = v2[i]   - v3[i];
    v1[i+1] = v2[i+1] - v3[i+1];
    v1[i+2] = v2[i+2] - v3[i+2];
    v1[i+3] = v2[i+3] - v3[i+3];
}


/****************
 * NCS functions.
 *
 * These are all designed on vectors with 64 float4 elements
 * and lid values in the range (0 - 15).
 *
 ****************/

__device__ void calcLLGradient(float4 *u,
                               float4 *data,
		               float4 *gamma,
		               float4 *gradient,
		               int lid)
{
    int i = lid*4;
    float4 t1;
    float4 t2;

    t1 = data[i] + gamma[i];
    t2.x = fmax(u[i].x + gamma[i].x, FITMIN);
    t2.y = fmax(u[i].y + gamma[i].y, FITMIN);
    t2.z = fmax(u[i].z + gamma[i].z, FITMIN);
    t2.w = fmax(u[i].w + gamma[i].w, FITMIN);
    gradient[i] = make_float4(1.0f, 1.0f, 1.0f, 1.0f) - t1/t2;

    i += 1;
    t1 = data[i] + gamma[i];
    t2.x = fmax(u[i].x + gamma[i].x, FITMIN);
    t2.y = fmax(u[i].y + gamma[i].y, FITMIN);
    t2.z = fmax(u[i].z + gamma[i].z, FITMIN);
    t2.w = fmax(u[i].w + gamma[i].w, FITMIN);
    gradient[i] = make_float4(1.0f, 1.0f, 1.0f, 1.0f) - t1/t2;

    i += 1;
    t1 = data[i] + gamma[i];
    t2.x = fmax(u[i].x + gamma[i].x, FITMIN);
    t2.y = fmax(u[i].y + gamma[i].y, FITMIN);
    t2.z = fmax(u[i].z + gamma[i].z, FITMIN);
    t2.w = fmax(u[i].w + gamma[i].w, FITMIN);
    gradient[i] = make_float4(1.0f, 1.0f, 1.0f, 1.0f) - t1/t2;

    i += 1;
    t1 = data[i] + gamma[i];
    t2.x = fmax(u[i].x + gamma[i].x, FITMIN);
    t2.y = fmax(u[i].y + gamma[i].y, FITMIN);
    t2.z = fmax(u[i].z + gamma[i].z, FITMIN);
    t2.w = fmax(u[i].w + gamma[i].w, FITMIN);
    gradient[i] = make_float4(1.0f, 1.0f, 1.0f, 1.0f) - t1/t2;    
}

__device__ void calcLogLikelihood(float *w1,
                                  float4 *u,
		                  float4 *data,
		                  float4 *gamma,
		                  int lid)
{
    int i = lid*4;
    float4 sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    float4 t1;
    float4 t2;

    t1 = data[i] + gamma[i];
    t2.x = log(fmax(u[i].x + gamma[i].x, FITMIN));
    t2.y = log(fmax(u[i].y + gamma[i].y, FITMIN));
    t2.z = log(fmax(u[i].z + gamma[i].z, FITMIN));
    t2.w = log(fmax(u[i].w + gamma[i].w, FITMIN));
    sum += u[i] - t1*t2;

    i += 1;
    t1 = data[i] + gamma[i];
    t2.x = log(fmax(u[i].x + gamma[i].x, FITMIN));
    t2.y = log(fmax(u[i].y + gamma[i].y, FITMIN));
    t2.z = log(fmax(u[i].z + gamma[i].z, FITMIN));
    t2.w = log(fmax(u[i].w + gamma[i].w, FITMIN));
    sum += u[i] - t1*t2;

    i += 1;
    t1 = data[i] + gamma[i];
    t2.x = log(fmax(u[i].x + gamma[i].x, FITMIN));
    t2.y = log(fmax(u[i].y + gamma[i].y, FITMIN));
    t2.z = log(fmax(u[i].z + gamma[i].z, FITMIN));
    t2.w = log(fmax(u[i].w + gamma[i].w, FITMIN));
    sum += u[i] - t1*t2;

    i += 1;
    t1 = data[i] + gamma[i];
    t2.x = log(fmax(u[i].x + gamma[i].x, FITMIN));
    t2.y = log(fmax(u[i].y + gamma[i].y, FITMIN));
    t2.z = log(fmax(u[i].z + gamma[i].z, FITMIN));
    t2.w = log(fmax(u[i].w + gamma[i].w, FITMIN));
    sum += u[i] - t1*t2;

    w1[lid] = sum.x + sum.y + sum.z + sum.w;

    __syncthreads();
    
    if(lid == 0){
    	for(i=1; i<16; i++){
	   w1[0] += w1[i];
	}
    }

    __syncthreads();        
}

/* 
 * Use inverse FFT optimization.
 *
 * Notes:
 *
 *  1. Only gives correct values for valid OTFs, meaning
 *     that they are the FT of a realistic PSF.
 *
 *  2. The u_fft_r and u_fft_c parameters must also be the
 *     FT of a real valued image.
 */
__device__ void calcNCGradientIFFT(float4 *w1,
                                  float4 *w2,
		                  float4 *w3,
                                  float4 *u_fft_r, 
                                  float4 *u_fft_c, 
                                  float4 *otf_mask_sqr,
                                  float4 *gradient,
			          int lid)
{
    int i = lid*4;
    
    float4 t1;

    t1 = 2.0f*otf_mask_sqr[i];
    w1[i] = t1*u_fft_r[i];
    w2[i] = t1*u_fft_c[i];

    i += 1;
    t1 = 2.0f*otf_mask_sqr[i];
    w1[i] = t1*u_fft_r[i];
    w2[i] = t1*u_fft_c[i];

    i += 1;
    t1 = 2.0f*otf_mask_sqr[i];
    w1[i] = t1*u_fft_r[i];
    w2[i] = t1*u_fft_c[i];

    i += 1;
    t1 = 2.0f*otf_mask_sqr[i];
    w1[i] = t1*u_fft_r[i];
    w2[i] = t1*u_fft_c[i];

    __syncthreads();
    
    ifft_16x16_wg16(w1, w2, gradient, w3, lid);
}

__device__ void calcNoiseContribution(float *w1,
                                      float4 *u_fft_r,
			              float4 *u_fft_c,
			              float4 *otf_mask_sqr,
			              int lid)
{
    int i = lid*4;
    float4 sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    float4 t1;
    
    t1 = u_fft_r[i]*u_fft_r[i] + u_fft_c[i]*u_fft_c[i];
    sum += t1*otf_mask_sqr[i];

    i += 1;
    t1 = u_fft_r[i]*u_fft_r[i] + u_fft_c[i]*u_fft_c[i];
    sum += t1*otf_mask_sqr[i];

    i += 1;
    t1 = u_fft_r[i]*u_fft_r[i] + u_fft_c[i]*u_fft_c[i];
    sum += t1*otf_mask_sqr[i];

    i += 1;
    t1 = u_fft_r[i]*u_fft_r[i] + u_fft_c[i]*u_fft_c[i];
    sum += t1*otf_mask_sqr[i];

    w1[lid] = sum.x + sum.y + sum.z + sum.w;

    __syncthreads();
    
    if(lid == 0){
    	for(i=1; i<16; i++){
	   w1[0] += w1[i];
	}
	w1[0] = w1[0] * (1.0f/(4.0f*PSIZE));
    }

    __syncthreads();        
}


/******************
 * L-BFGS functions.
 ******************/

__device__ void converged(int *w1,
                          float *w2,
	                  float4 *x,
	                  float4 *g,
	                  int lid)
{
    float xnorm;
    float gnorm;
    
    vecnorm(w2, x, lid); 
    if (lid == 0){
        xnorm = fmax(w2[0], 1.0f);
    }

    __syncthreads();
    
    vecnorm(w2, g, lid);
    if (lid == 0){
        gnorm = w2[0];
	if ((gnorm/xnorm) > EPSILON){
	    w1[0] = 0;
        }
        else{
            w1[0] = 1;
        }
    }

    __syncthreads();
}


/****************
 * Kernels.
 ****************/

/*
 * Run NCS noise reduction on sub-regions.
 * 
 * Note: Any zero or negative values in the sub-regions should be
 *       set to a small positive value like 1.0.
 *
 * data_in - Sub-region data in e-.
 * g_gamma - Sub-region CMOS variance in units of e-^2.
 * otf_mask - 16 x 16 array containing the OTF mask.
 * data_out - Storage for noise corrected sub-regions.
 * iterations - Number of L-BFGS solver iterations.
 * status - Status of the solution (good, failed because of X).
 * alpha - NCS alpha term.
 */
extern "C" __global__ void ncsReduceNoise(float4 *data_in,
                                          float4 *g_gamma,
                                          float4 *otf_mask,
                                          float4 *data_out,
                                          int *g_iterations,
                                          int *g_status,
                                          float alpha)
{
    int gid = blockIdx.x;
    int lid = threadIdx.x;

    int i = lid*4;
    int i_g = gid*PSIZE;

    int j,k;
    
    __shared__ int bound;
    __shared__ int ci;
    __shared__ int searching;
  
    __shared__ float beta;
    __shared__ float cost;
    __shared__ float cost_p;
    __shared__ float step;
    __shared__ float t1;
    __shared__ float ys_c0;
    __shared__ float yy;

    __shared__ int w1_i[ASIZE];
    __shared__ float w1_f[ASIZE];
    
    __shared__ float a[M];
    __shared__ float ys[M];

    __shared__ float4 data[PSIZE];
    __shared__ float4 gamma[PSIZE];
    __shared__ float4 g_p[PSIZE];
    __shared__ float4 gradient[PSIZE];
    __shared__ float4 otf_mask_sqr[PSIZE];
    __shared__ float4 srch_dir[PSIZE];
    __shared__ float4 u_c[PSIZE];
    __shared__ float4 u_p[PSIZE];
    __shared__ float4 u_r[PSIZE]; 
    __shared__ float4 u_fft_r[PSIZE]; 
    __shared__ float4 u_fft_c[PSIZE];

    __shared__ float4 w1_f4[PSIZE];
    __shared__ float4 w2_f4[PSIZE];
    __shared__ float4 w3_f4[PSIZE];
    __shared__ float4 w4_f4[PSIZE];
    
    __shared__ float4 s[M][PSIZE];
    __shared__ float4 y[M][PSIZE];

    /* Initialization. */
    for (j=0; j<4; j++){
    	k = i+j;
        data[k] = data_in[i_g+k];
        gamma[k] = g_gamma[i_g+k];
        otf_mask_sqr[k] = otf_mask[k] * otf_mask[k];
        u_r[k] = data_in[i_g+k];
	u_c[k] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    }

    __syncthreads();

    /* Calculate initial state. */
    
    /* Cost. */
    fft_16x16_wg16(u_r, u_c, u_fft_r, u_fft_c, lid);

    calcLogLikelihood(w1_f, u_r, data, gamma, lid);
    if (lid == 0){
        cost = w1_f[0];
    }

    __syncthreads();

    calcNoiseContribution(w1_f, u_fft_r, u_fft_c, otf_mask_sqr, lid);
    if (lid == 0){
        cost += alpha*w1_f[0];
    }

    __syncthreads();
    
    /* Gradient. */
    calcLLGradient(u_r, data, gamma, gradient, lid);
    calcNCGradientIFFT(w1_f4, w2_f4, w3_f4, u_fft_r, u_fft_c, otf_mask_sqr, w4_f4, lid);
    vecfmaInplace(gradient, w4_f4, alpha, lid);

    __syncthreads();
    
    /* Check if we've already converged. */
    converged(w1_i, w1_f, u_r, gradient, lid);
    if (w1_i[0]){
        if (lid == 0){
            g_iterations[gid] = 1;
            g_status[gid] = SUCCESS;
	}
	for (j=0; j<4; j++){
            k = i+j;
    	    data_out[i_g+k] = u_r[k];
        }
	return;
    }
    
    /* Initial search direction. */
    vecnorm(w1_f, gradient, lid);
    if (lid == 0){
        step = 1.0/w1_f[0];
    }
    vecncopy(srch_dir, gradient, lid);

    __syncthreads();
    
    /* Start search. */
    for (k=1; k<(MAXITERS+1); k++){
    
        /* 
         * Line search. 
         *
         * This checks the Armijo rule/condition.
         * https://en.wikipedia.org/wiki/Wolfe_conditions
         */
	vecdot(w1_f, srch_dir, gradient, lid);
	if (lid == 0){
            t1 = C_1 * w1_f[0];
        }

        __syncthreads();
	
        if (t1 > 0.0){
            /* Increasing gradient. Minimization failed. */
            if (lid == 0){
                g_iterations[gid] = k;
                g_status[gid] = INCREASING_GRADIENT;
	    }
	    for (j=0; j<4; j++){
                k = i+j;
    	        data_out[i_g+k] = u_r[k];
            }
	    return;
        }
        
        /* Store current cost, u and gradient. */
	if (lid == 0){
            cost_p = cost;
	}
        veccopy(u_p, u_r, lid);
        veccopy(g_p, gradient, lid);
	
	/* Search for a good step size. */
	if (lid == 0){
            searching = 1;
	}

        __syncthreads();
	
        while(searching){
        
            /* Move in search direction. */
            vecfma(u_r, srch_dir, u_p, step, lid);

            __syncthreads();

            fft_16x16_wg16(u_r, u_c, u_fft_r, u_fft_c, lid);

            /* Calculate new cost. */
            calcLogLikelihood(w1_f, u_r, data, gamma, lid);
            if (lid == 0){
                cost = w1_f[0];
            }

            __syncthreads();

            calcNoiseContribution(w1_f, u_fft_r, u_fft_c, otf_mask_sqr, lid);
            if (lid == 0){
                cost += alpha*w1_f[0];
            }

            __syncthreads();
            
            /* Armijo condition. */
            if (cost <= (cost_p + t1*step)){
	        if (lid == 0){
                    searching = 0;
		}

                __syncthreads();
		
            }
            else{
	        if (lid == 0){ 
                    step = STEPM*step;
		}

                __syncthreads();
		
                if (step < MIN_STEP){
                    /* 
                     * Reached minimum step size. Minimization failed. 
                     * Return the last good u values.
                     */
                    if (lid == 0){
                        g_iterations[gid] = k+1;
                        g_status[gid] = MINIMUM_STEP;
	            }
		    for (j=0; j<4; j++){
                         k = i+j;
    	                 data_out[i_g+k] = u_p[k];
                    }
                    return;
                }
            }
        }
        
        /* Calculate new gradient. */
	calcLLGradient(u_r, data, gamma, gradient, lid);
        calcNCGradientIFFT(w1_f4, w2_f4, w3_f4, u_fft_r, u_fft_c, otf_mask_sqr, w4_f4, lid);
        vecfmaInplace(gradient, w4_f4, alpha, lid);

        __syncthreads();
    
        /* Convergence check. */
        converged(w1_i, w1_f, u_r, gradient, lid);
        if (w1_i[0]){
            if (lid == 0){
                g_iterations[gid] = k+1;
                g_status[gid] = SUCCESS;
	    }
            for (j=0; j<4; j++){
                k = i+j;
    	        data_out[i_g+k] = u_r[k];
            }
	    return;
        }
        
        /*
	 * Machine precision check.
	 *
	 * This is probably not an actual failure, we just ran out of digits. Reaching
	 * this state has a cost so we want to know if this is happening a lot.
	 */
	vecisEqual(w1_i, u_r, u_p, lid);
        if (w1_i[0]){
	    if (lid == 0){
                g_iterations[gid] = k+1;
                g_status[gid] = REACHED_MAXPRECISION;
            }
            for (j=0; j<4; j++){
                k = i+j;
    	        data_out[i_g+k] = u_r[k];
            }	    
            return;
        }
        
        /* L-BFGS calculation of new search direction. */
	if (lid == 0){
            ci = (k-1)%M;
	}

        __syncthreads();
	
        vecsub(s[ci], u_r, u_p, lid);
        vecsub(y[ci], gradient, g_p, lid);

        __syncthreads();

        vecdot(w1_f, s[ci], y[ci], lid);
	if (lid == 0){
            ys_c0 = w1_f[0];
            ys[ci] = 1.0/ys_c0;
	}

        __syncthreads(); 

        vecdot(w1_f, y[ci], y[ci], lid);
        if (lid == 0){
	    yy = 1.0/w1_f[0];
        }

        vecncopy(srch_dir, gradient, lid);

        if (lid == 0){
            bound = min(k, M);
	}

        __syncthreads();
	
        for(j=0; j<bound; j++){

            if (lid == 0){
	        ci = (k - j - 1)%M;
	    }
	    
            __syncthreads();

            vecdot(w1_f, s[ci], srch_dir, lid);
            if (lid == 0){
                a[ci] = w1_f[0]*ys[ci];
            }

            __syncthreads();

            vecfmaInplace(srch_dir, y[ci], -a[ci], lid);
        }
        
        vecscaleInplace(srch_dir, ys_c0*yy, lid);
        
        for(j=0; j<bound; j++){

            if (lid == 0){
	        ci = (k + j - bound)%M;
	    }

            __syncthreads();

            vecdot(w1_f, y[ci], srch_dir, lid);
	    if (lid == 0){
	        beta = w1_f[0]*ys[ci];
	    }

            __syncthreads();

            vecfmaInplace(srch_dir, s[ci], (a[ci] - beta), lid);
        }

        if (lid == 0){
            step = 1.0;
	}

        __syncthreads();
    }
    
    /* Reached maximum iterations. Minimization failed. */
    if (lid == 0){
        g_iterations[gid] = MAXITERS;
        g_status[gid] = REACHED_MAXITERS;
    }
    for (j=0; j<4; j++){
        k = i+j;
    	data_out[i_g+k] = u_r[k];
    }
}

